#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

std::vector<at::Tensor> maxmin_cuda_forward(
    at::Tensor input,
    int32_t axis) {

  return {input};
}

template <typename scalar_t>
__global__ void maxmin_cuda_forward_kernel(
    const scalar_t* __restrict__ gates,
    const scalar_t* __restrict__ old_cell,
    scalar_t* __restrict__ new_h,
    scalar_t* __restrict__ new_cell,
    scalar_t* __restrict__ input_gate,
    scalar_t* __restrict__ output_gate,
    scalar_t* __restrict__ candidate_cell,
    size_t state_size) {
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  const int index = blockIdx.y * state_size + column;
  const int gates_row = blockIdx.y * (state_size * 3);
  
  }
}

std::vector<at::Tensor> maxmin_cuda_backward(
    at::Tensor input,
    at::Tensor grad,
    int32_t axis) {

  return {grad};
}

template <typename scalar_t>
__global__ void maxmin_cuda_backward_kernel(
    const scalar_t* __restrict__ gates,
    const scalar_t* __restrict__ old_cell,
    scalar_t* __restrict__ new_h,
    scalar_t* __restrict__ new_cell,
    scalar_t* __restrict__ input_gate,
    scalar_t* __restrict__ output_gate,
    scalar_t* __restrict__ candidate_cell,
    size_t state_size) {
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  const int index = blockIdx.y * state_size + column;
  const int gates_row = blockIdx.y * (state_size * 3);
  
  }
}

